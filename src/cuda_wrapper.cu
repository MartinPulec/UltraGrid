/**
 * @file   cuda_wrapper.h
 * @author Martin Pulec     <pulec@cesnet.cz>
 *
 * @brief  This file contais wrapper around CUDA functions.
 */
/*
 * Copyright (c) 2013 CESNET z.s.p.o.
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, is permitted provided that the following conditions
 * are met:
 *
 * 1. Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *
 * 3. Neither the name of CESNET nor the names of its contributors may be
 *    used to endorse or promote products derived from this software without
 *    specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE AUTHORS AND CONTRIBUTORS
 * "AS IS" AND ANY EXPRESSED OR IMPLIED WARRANTIES, INCLUDING,
 * BUT NOT LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY
 * AND FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO
 * EVENT SHALL THE AUTHORS OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT,
 * INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 * (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION)
 * HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
 * CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR
 * OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE,
 * EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include "cuda_wrapper.h"

#include "hip/hip_runtime.h"
#include <stdlib.h>

typedef void *cuda_wrapper_stream_t;

static inline int map_cuda_error(hipError_t cuda_error) {
        return (int) cuda_error;
};

static inline const char * map_error_string(int error) {

        return hipGetErrorString((hipError_t) error);
};

static inline enum hipMemcpyKind map_cuda_memcpy_kind(int our_kind) {
        struct kind_mapping {
                enum hipMemcpyKind kind;
                int our_kind;
        };
        struct kind_mapping mapping[] = {
                { hipMemcpyHostToDevice, CUDA_WRAPPER_MEMCPY_HOST_TO_DEVICE },
                { hipMemcpyDeviceToHost, CUDA_WRAPPER_MEMCPY_DEVICE_TO_HOST },
                { hipMemcpyDeviceToDevice, CUDA_WRAPPER_MEMCPY_DEVICE_TO_DEVICE },
        };

        int i;
        for (i = 0; i < sizeof(mapping)/sizeof(struct kind_mapping); ++i) {
                if (our_kind == mapping[i].our_kind) {
                        return mapping[i].kind;
                }
        }

        abort(); // should not reach here
};

CUDA_DLL_API int cuda_wrapper_free(void *buffer)
{
        return map_cuda_error(hipFree(buffer));
}

CUDA_DLL_API int cuda_wrapper_free_host(void *buffer)
{
        return map_cuda_error(hipHostFree(buffer));
}

CUDA_DLL_API int cuda_wrapper_host_alloc(void **pHost, size_t size, unsigned int flags)
{
        return map_cuda_error(hipHostAlloc(pHost, size, flags));
}

CUDA_DLL_API int cuda_wrapper_malloc(void **buffer, size_t data_len)
{
        return map_cuda_error(hipMalloc(buffer, data_len));
}

CUDA_DLL_API int cuda_wrapper_malloc_host(void **buffer, size_t data_len)
{
        return map_cuda_error(hipHostMalloc(buffer, data_len));
}

CUDA_DLL_API int cuda_wrapper_memcpy(void *dst, const void *src,
                size_t count, int kind)
{
        return map_cuda_error(
                        hipMemcpy(dst, src, count,
                                map_cuda_memcpy_kind(kind)));
}

CUDA_DLL_API const char *cuda_wrapper_last_error_string(void)
{
        return hipGetErrorString(hipGetLastError());
}

CUDA_DLL_API int cuda_wrapper_get_last_error(void)
{
        return map_cuda_error(hipGetLastError());
}

CUDA_DLL_API const char *cuda_wrapper_get_error_string(int error)
{
        return map_error_string(error);
}

CUDA_DLL_API int cuda_wrapper_set_device(int index)
{
        return map_cuda_error(
                        hipSetDevice(index));
}

