#include "hip/hip_runtime.h"
/* 
 * Copyright (c) 2011, Martin Srom
 * All rights reserved.
 * 
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 * 
 *     * Redistributions of source code must retain the above copyright
 *       notice, this list of conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright
 *       notice, this list of conditions and the following disclaimer in the
 *       documentation and/or other materials provided with the distribution.
 * 
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
 * ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE
 * LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
 * CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
 * SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
 * INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
 * CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
 * ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
 * POSSIBILITY OF SUCH DAMAGE.
 */

#include "register.h"
#include "common.h"

#define THREAD_CBLK_SIZE 64

/** MQ-Coder lookup table */
__constant__ struct mqc_cxstate
d_mqc_gpu_register_table[mqc_table_size];

/**
 * Perform byte out procedure
 * 
 * @param c  Code register
 * @param ct  Free space in code register
 * @param bp  Output byte buffer
 */
__device__ inline void
mqc_gpu_register_byte_out(uint32_t & c, uint8_t & ct, uint8_t* & bp)
{
    if ( *bp == 0xff ) {
        bp++;
        *bp = c >> 20;
        c &= 0xfffff;
        ct = 7;
    } else {
        if ( (c & 0x8000000) == 0 ) {
            bp++;
            *bp = c >> 19;
            c &= 0x7ffff;
            ct = 8;
        } else {
            (*bp)++;
            if ( *bp == 0xff ) {
                c &= 0x7ffffff;
                bp++;
                *bp = c >> 20;
                c &= 0xfffff;
                ct = 7;
            } else {
                bp++;
                *bp = c >> 19;
                c &= 0x7ffff;
                ct = 8;
            }
        }
    }
}

/**
 * Perform code MPS procedure
 * 
 * @param a  Interval register
 * @param c  Code register
 * @param ct  Free space in code register
 * @param bp  Output byte buffer
 * @param ctx  Reference to current context
 * @param state  Current context state
 */
__device__ inline void
mqc_gpu_register_code_mps(uint32_t & a, uint32_t & c, uint8_t & ct, uint8_t* & bp, uint8_t* & ctx, struct mqc_cxstate* state)
{
    int qeval = state->qeval;
    a -= qeval;
    if ( (a & 0x8000) == 0 ) {
        if (a < qeval) {
            a = qeval;
        } else {
            c += qeval;
        }
        *ctx = state->nmps;

        while ( (a & 0x8000) == 0 ) {
            a <<= 1;
            c <<= 1;
            ct--;
            if (ct == 0) {
                mqc_gpu_register_byte_out(c,ct,bp);
            }
        }
    } else {
        c += qeval;
    }
}

/**
 * Perform code LPS procedure
 * 
 * @param a  Interval register
 * @param c  Code register
 * @param ct  Free space in code register
 * @param bp  Output byte buffer
 * @param ctx  Reference to current context
 * @param state  Current context state
 */
__device__ inline void
mqc_gpu_register_code_lps(uint32_t & a, uint32_t & c, uint8_t & ct, uint8_t* & bp, uint8_t* & ctx, struct mqc_cxstate* state)
{
    uint32_t qeval =  state->qeval;
    a -= qeval;
    if ( a < qeval ) {
        c += qeval;
    } else {
        a = qeval;
    }

    *ctx = state->nlps;

    while ( (a & 0x8000) == 0 ) {
        a <<= 1;
        c <<= 1;
        ct--;
        if (ct == 0) {
            mqc_gpu_register_byte_out(c,ct,bp);
        }
    }
}

/**
 * Perform flush last bytes procedure
 * 
 * @param a  Interval register
 * @param c  Code register
 * @param ct  Free space in code register
 * @param bp  Output byte buffer
 */
__device__ inline void
mqc_gpu_register_flush(uint32_t & a, uint32_t & c, uint8_t & ct, uint8_t* & bp)
{
    uint64_t tempc = c + a;
    c |= 0xffff;
    if ( c >= tempc ) {
        c -= 0x8000;
    }
    c <<= ct;
    mqc_gpu_register_byte_out(c,ct,bp);
    c <<= ct;
    mqc_gpu_register_byte_out(c,ct,bp);
    if ( *bp != 0xff ) {
        bp++;
    }
    c = c;
    ct = ct;
}

/**
 * Kernel that performs MQ-Encoding for one block
 * 
 * @param d_cblk  Array of code-blocks in device memory
 * @param cblk_count  Count of code-blocks
 * @param d_cxd  Array of input CX,D pairs in device memory
 * @param d_byte  Array of output bytes in device memory
 */
__global__ void
mqc_gpu_register_encode_kernel(struct j2k_cblk* d_cblk, int cblk_count, uint8_t* d_cxd, uint8_t* d_byte)
{
    // Get and check block index
    int block_index = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x;
    if ( block_index >= cblk_count )
        return;

    // Get block of CX,D pairs
    struct j2k_cblk* block = &d_cblk[block_index];
    
    // Output byte stream
    uint8_t* start = &d_byte[block->byte_index];

    // Init variables
    uint32_t a = 0x8000;
    uint32_t c = 0;
    uint8_t ct = 12;
    uint8_t* bp = start - 1;
    if ( *bp == 0xff ) {
        ct = 13;
    }

    // Init contexts
    uint8_t ctxs[19];
    mqc_gpu_reset_cxstate(ctxs);
   
    // Code CX,D pairs
    int cxd_begin = block->cxd_index;
    int cxd_count = cxd_begin + block->cxd_count;
    int cxd_index = cxd_begin;
    while ( cxd_index < cxd_count ) {
        uint8_t cxd = d_cxd[cxd_index]; 
        uint8_t* ctx = &ctxs[mqc_gpu_cxd_get_cx(cxd)];
        struct mqc_cxstate* state = &d_mqc_gpu_register_table[*ctx];
        if ( state->mps == mqc_gpu_cxd_get_d(cxd) ) {
            mqc_gpu_register_code_mps(a,c,ct,bp,ctx,state);
        } else {
            mqc_gpu_register_code_lps(a,c,ct,bp,ctx,state);
        }
        cxd_index++;
    }

    // Flush last bytes
    mqc_gpu_register_flush(a,c,ct,bp); 

    // Set output byte count
    block->byte_count = bp - start;
}

/** Documented at declaration */
void*
mqc_gpu_register_create(struct mqc_configuration * configuration)
{
    // Init lookup table
    mqc_gpu_init_table("d_mqc_gpu_register_table");

    // Configure L1
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(mqc_gpu_register_encode_kernel), hipFuncCachePreferL1);
    
    return 0;
}

/** Documented at declaration */
int
mqc_gpu_register_encode(void* mqc, struct j2k_cblk* d_cblk, int cblk_count, unsigned char * d_cxd, unsigned char * d_byte)
{
    dim3 dim_grid;
    dim_grid.x = cblk_count / THREAD_CBLK_SIZE + 1;
    if ( dim_grid.x > CUDA_MAXIMUM_GRID_SIZE ) {
        dim_grid.y = dim_grid.x / CUDA_MAXIMUM_GRID_SIZE + 1;
        dim_grid.x = CUDA_MAXIMUM_GRID_SIZE;
    }

    // Run kernel encode
    mqc_gpu_register_encode_kernel<<<dim_grid,THREAD_CBLK_SIZE>>>(d_cblk, cblk_count, d_cxd, d_byte);
    hipError_t cuerr = hipDeviceSynchronize();
    if ( cuerr != hipSuccess ) {
        std::cerr << "Kernel encode failed: " << hipGetErrorString(cuerr) << std::endl;
        return -1;
    }
    return 0;
}

/** Documented at declaration */
int
mqc_gpu_register_destroy(void* mqc)
{
    return 0;
}

