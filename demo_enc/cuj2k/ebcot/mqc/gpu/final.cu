#include "hip/hip_runtime.h"
/* 
 * Copyright (c) 2011, Martin Srom
 * All rights reserved.
 * 
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 * 
 *     * Redistributions of source code must retain the above copyright
 *       notice, this list of conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright
 *       notice, this list of conditions and the following disclaimer in the
 *       documentation and/or other materials provided with the distribution.
 * 
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
 * ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE
 * LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
 * CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
 * SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
 * INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
 * CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
 * ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
 * POSSIBILITY OF SUCH DAMAGE.
 */

#include "final.h"
#include "common.h"

/** MQ-Coder lookup table */
__constant__ struct mqc_cxstate
d_mqc_gpu_final_table[mqc_table_size];

/**
 * Perform byte out procedure
 * 
 * @param c  Code register
 * @param ct  Free space in code register
 * @param bp  Output byte buffer
 */
__device__ inline void
mqc_gpu_final_byte_out(uint32_t & c, uint8_t & ct, uint8_t* & bp)
{
    if ( *bp == 0xff ) {
        bp++;
        *bp = c >> 20;
        c &= 0xfffff;
        ct = 7;
    } else {
        if ( (c & 0x8000000) == 0 ) {
            bp++;
            *bp = c >> 19;
            c &= 0x7ffff;
            ct = 8;
        } else {
            (*bp)++;
            if ( *bp == 0xff ) {
                c &= 0x7ffffff;
                bp++;
                *bp = c >> 20;
                c &= 0xfffff;
                ct = 7;
            } else {
                bp++;
                *bp = c >> 19;
                c &= 0x7ffff;
                ct = 8;
            }
        }
    }
}

/**
 * Perform code MPS procedure
 * 
 * @param a  Interval register
 * @param c  Code register
 * @param ct  Free space in code register
 * @param bp  Output byte buffer
 * @param ctx  Reference to current context
 * @param state  Current context state
 */
__device__ inline void
mqc_gpu_final_code_mps(uint32_t & a, uint32_t & c, uint8_t & ct, uint8_t* & bp, uint8_t & ctx, struct mqc_cxstate* state)
{
    int qeval =  state->qeval;
    a -= qeval;
    if ( (a & 0x8000) == 0 ) {
        if (a < qeval) {
            a = qeval;
        } else {
            c += qeval;
        }
        ctx = state->nmps;

        a <<= 1;
        c <<= 1;
        ct--;
        if (ct == 0) {
            mqc_gpu_final_byte_out(c, ct, bp);
        }
    } else {
        c += qeval;
    }
}

/**
 * Perform code LPS procedure
 * 
 * @param a  Interval register
 * @param c  Code register
 * @param ct  Free space in code register
 * @param bp  Output byte buffer
 * @param ctx  Reference to current context
 * @param state  Current context state
 */
__device__ inline void
mqc_gpu_final_code_lps(uint32_t & a, uint32_t & c, uint8_t & ct, uint8_t* & bp, uint8_t & ctx, struct mqc_cxstate* state)
{
    int qeval = state->qeval;
    a -= qeval;
    if ( a < qeval ) {
        c += qeval;
    } else {
        a = qeval;
    }

    ctx = state->nlps;

    int ns = __clz(a) - (sizeof(uint32_t) * 8 - 16);
    a = a << ns;
    if ( ct > ns ) {
        c = c << ns;
        ct = ct - ns;
    } else {
        c = c << ct;
        ns = ns - ct;
        mqc_gpu_final_byte_out(c, ct, bp);
        if ( ct > ns ) {
            c = c << ns;
            ct = ct - ns;
        } else {
            c = c << ct;
            ns = ns - ct;
            mqc_gpu_final_byte_out(c, ct, bp);
            ct = ct - ns;
            c = c << ns;
        }
    }
}

/**
 * Perform flush last bytes procedure
 * 
 * @param a  Interval register
 * @param c  Code register
 * @param ct  Free space in code register
 * @param bp  Output byte buffer
 */
__device__ inline void
mqc_gpu_final_flush(uint32_t & a, uint32_t & c, uint8_t & ct, uint8_t* & bp)
{
    unsigned int tempc = c + a;
    c |= 0xffff;
    if ( c >= tempc ) {
        c -= 0x8000;
    }
    c <<= ct;
    mqc_gpu_final_byte_out(c,ct,bp);
    c <<= ct;
    mqc_gpu_final_byte_out(c,ct,bp);
    if ( *bp != 0xff ) {
        bp++;
    }
    c = c;
    ct = ct;
}

/**
 * Encode one CX,D pair
 * 
 * @param cxd  CX,D pair
 * @param a  Interval register
 * @param c  Code register
 * @param ct  Free space in code register
 * @param bp  Output byte buffer
 * @param cxstate  Context states
 * @param d_trunc_size  Byte sizes for truncation points
 * @return true if continue in coding, otherwise false
 */
__device__ inline void
mqc_gpu_final_encode_symbol(uint8_t cxd, uint32_t & a, uint32_t & c, uint8_t & ct, uint8_t* & start, uint8_t* & bp, uint8_t (& cxstate)[19],
                            int & cxd_index, int & cxd_count, int & pass_count, unsigned int* d_trunc_size)
{
    // Skip coding of end of pass CX,D
    if ( mqc_gpu_cxd_is_pass_end(cxd) ) {
        // Increment processed pass
        pass_count++;
        d_trunc_size[pass_count] = (bp - start) + 3;
    }
    // Perform CX,D coding of MPS or LPS
    else {
        uint8_t & ctx = cxstate[mqc_gpu_cxd_get_cx(cxd)];
        struct mqc_cxstate* state = &d_mqc_gpu_final_table[ctx];
        if ( state->mps == mqc_gpu_cxd_get_d(cxd) ) {
            mqc_gpu_final_code_mps(a, c, ct, bp, ctx, state);
        } else {
            mqc_gpu_final_code_lps(a, c, ct, bp, ctx, state);
        }
    }
}

/**
 * Kernel that performs MQ-Encoding for one block
 * 
 * @param d_cblk  Array of code-blocks in device memory
 * @param cblk_count  Count of code-blocks
 * @param d_cxd  Array of input CX,D pairs in device memory
 * @param d_byte  Array of output bytes in device memory
 * @param d_trunc_size  Array of sizes for truncation points (after each pass)
 */
template<
    // Thread Work Count (how many threads in thread block will process code-blocks)
    unsigned int threadWorkCount,
    // Thread Per Count (how many threads will be one group of one working thread, 
    // 1 means only working threads, 2 means separation by 1 thread, etc.)
    unsigned int threadPerCount,
    // Data type used for batch loading (ie. unsigned char, int, double, etc.)
    class cxdLoadType,
    // How many values of specified data type will be used for one batch load
    unsigned int cxdLoadCount
>
__global__ void
mqc_gpu_final_encode_kernel(struct j2k_cblk* d_cblk, int cblk_count, unsigned char* d_cxd, unsigned char * d_byte, unsigned int* d_trunc_sizes)
{
    // Get and check block index
    int cblk_index = (blockIdx.y * gridDim.x + blockIdx.x) * threadWorkCount + threadIdx.x / threadPerCount;
    if ( cblk_index >= cblk_count )
        return;

    // Thread index in count
    int thread_index = threadIdx.x % threadPerCount;

    // Is this thread working (not working threads do nothing)
    bool work_thread = (thread_index) == 0;
    if ( work_thread == false )
        return;

    // Get code-block for working thread to process
    struct j2k_cblk* cblk = &d_cblk[cblk_index];

    // CX,D info
    int cxd_begin = cblk->cxd_index;
    int cxd_count = cxd_begin + cblk->cxd_count;
    int cxd_index = cxd_begin;

    // Output byte stream
    uint8_t* start = &d_byte[cblk->byte_index];

    // Init variables
    uint32_t a = 0x8000;
    uint32_t c = 0;
    uint8_t ct = 12;
    uint8_t* bp = start - 1;
    uint8_t cxstate[19];
    mqc_gpu_reset_cxstate(cxstate);
    // Reset first byte (will be checked for 0xFF in byte out procedure)
    bp[0] = 0;
    int pass_count = 0;
    
    // Output truncation point size pointer (first size is 0 == no bytes at all)
    unsigned int* d_trunc_size = &d_trunc_sizes[cblk->trunc_index];
    d_trunc_size[0] = 0;
    
    if ( sizeof(cxdLoadType) == 1 && cxdLoadCount == 1 ) {
        // Encode CX,D
        for ( cxd_index = cxd_index; cxd_index < cxd_count; cxd_index++ ) {
            uint8_t cxd = d_cxd[cxd_index];
            mqc_gpu_final_encode_symbol(cxd, a, c, ct, start, bp, cxstate, cxd_index, cxd_count, pass_count, d_trunc_size);
         }
    } else {
        // Get count of CX,D for align
        int align_count = cxd_index % sizeof(cxdLoadType);
        if ( align_count > 0 ) {
            // Make differ
            align_count = cxd_index + sizeof(cxdLoadType) - align_count;
            // Check count
            if ( align_count > cxd_count )
                align_count = cxd_count;
            // Encode align symbols
            for ( cxd_index = cxd_index; cxd_index < align_count; cxd_index++ ) {
                uint8_t cxd = d_cxd[cxd_index];
                mqc_gpu_final_encode_symbol(cxd, a, c, ct, start, bp, cxstate, cxd_index, cxd_count, pass_count, d_trunc_size);
            }
        }

        // Encode
        while ( cxd_index < cxd_count ) {
            // Init count
            int count = sizeof(cxdLoadType) * cxdLoadCount;
            if ( (cxd_index + count) >= cxd_count ) {
                count = cxd_count - cxd_index;
            }

            // Load CX,D by load type
            cxdLoadType cxd_data[cxdLoadCount];
            for ( int index = 0; index < cxdLoadCount; index++ )
                cxd_data[index] = reinterpret_cast<cxdLoadType*>(&d_cxd[cxd_index])[index];

            // Encode CX,D
            for ( int index = 0; index < count; index++ ) {
                uint8_t cxd = reinterpret_cast<uint8_t*>(&cxd_data)[index];
                mqc_gpu_final_encode_symbol(cxd, a, c, ct, start, bp, cxstate, cxd_index, cxd_count, pass_count, d_trunc_size);
            }
        
            cxd_index += count;
        }
    }
    
    // Flush last bytes
    mqc_gpu_final_flush(a, c, ct, bp);

    // Set output byte count and correct last truncation point
    cblk->byte_count = bp - start;
    d_trunc_size[pass_count] = bp - start;
    // Set processed pass count
    cblk->pass_count = pass_count;
    cblk->trunc_count = pass_count + 1; // one truncation is always at the begin (meaning that the codeblock is not coded at all)
}

/** MQ-Coder kernel type */
typedef void (*mqc_kernel)(struct j2k_cblk*, int, unsigned char*, unsigned char*, unsigned int*);

/** Thread Work Count (how many threads in thread block will process code-blocks) */
const int twc = 64;
/** Thread Per Count (how many threads will be one group of one working thread, 1 means only working threads, 2 means separation by 1 thread, etc.) */
const int tpc = 1;

/** Documented at declaration */
void*
mqc_gpu_final_create(struct mqc_configuration * configuration)
{
    // Init lookup table
    mqc_gpu_init_table("d_mqc_gpu_final_table");
    
    // Select kernel   TODO: version with/without truncation point tracking
    mqc_kernel mqc = NULL;
    mqc = &mqc_gpu_final_encode_kernel<twc, tpc, uint64_t, 16>;
    assert(mqc != NULL);

    // Configure L1
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(mqc), hipFuncCachePreferL1);
    
    return (void*)mqc;
}

/** Documented at declaration */
int
mqc_gpu_final_encode(void* mqc, struct j2k_cblk* d_cblk, int cblk_count, unsigned char * d_cxd, unsigned char * d_byte, unsigned int * d_trunc_sizes, hipStream_t stream)
{
    // Calculate grid and block sizes
    int count = cblk_count / twc + 1;
    dim3 dim_grid;
    dim_grid.x = count;
    if ( dim_grid.x > CUDA_MAXIMUM_GRID_SIZE ) {
        dim_grid.x = CUDA_MAXIMUM_GRID_SIZE;
        dim_grid.y = count / CUDA_MAXIMUM_GRID_SIZE + 1;
    }
    dim3 dim_block(twc * tpc, 1);
    
    // Perform encoding
    ((mqc_kernel)mqc)<<<dim_grid, dim_block, 0, stream>>>(
        d_cblk,
        cblk_count,
        d_cxd,
        d_byte,
        d_trunc_sizes
    );
//     hipError_t cuerr = hipDeviceSynchronize();
//     if ( cuerr != hipSuccess ) {
//         std::cerr << "MQ-Coder Kernel encoding failed: " << hipGetErrorString(cuerr) << std::endl;
//         return -1;
//     }
    return 0;
}

/** Documented at declaration */
int
mqc_gpu_final_destroy(void* mqc)
{
    // De-configure L1
    hipFuncSetCacheConfig(((mqc_kernel)mqc), hipFuncCachePreferNone);
    
    return 0;
}

